#include "hip/hip_runtime.h"
//fir filtering via fft with cuda

//includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>


// includes, project
#include <hipfft/hipfft.h>
#include <cutil.h>

// Complex data type
typedef float Complex[2]; 
typedef float Real;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ 			void  ComplexPointwiseMulAndScale(Complex*, const Complex*, int, float);

// Filtering functions
void fftFilter(float* hsignal, float* kernel, int n)
{// filteredSignal will hold the convolution of signal and kernel, all of which have n elements.
	
    int memSizeReal    = sizeof(Real) * n;
	int memSizeComplex = sizeof(Complex) * (n/2+1);
 
    // Allocate device memory for signal
    Real* dSignal;
    CUDA_SAFE_CALL(hipMalloc((void**)&dSignal, memSizeReal));
    // Copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(dSignal, hSignal,memSizeReal,
                              hipMemcpyHostToDevice));

    // Allocate device memory for filter kernel signal, and transforms
    Real* dKernel;
	CUDA_SAFE_CALL(hipMalloc((void**)&dKernel, memSizeReal));

    // Copy host memory to device
    CUDA_SAFE_CALL(hipMemcpy(dKernel, hKernel, memSizeReal,
                              hipMemcpyHostToDevice));

	//allocate device memory for transforms
	Complex* dKernelTransform;
	Complex* dSignalTransform;
	CUDA_SAFE_CALL(hipMalloc((void**)&dKernelTransform, memSizeComplex));
	CUDA_SAFE_CALL(hipMalloc((void**)&dSignalTransform, memSizeComplex));
		
    // CUFFT plan
    hipfftHandle fPlan;//forward plan
	hipfftHandle rPlan;//reverse plan
	
    CUFFT_SAFE_CALL(hipfftPlan1d(&fPlan, n, HIPFFT_R2C, 1));
	CUFFT_SAFE_CALL(hipfftPlan1d(&rPlan, n, HIPFFT_C2R, 1));

    // Transform signal and kernel
	
    CUFFT_SAFE_CALL(hipfftExecR2C(fPlan, (hipfftReal *)dSignal, (hipfftComplex *)dSignalTransform));
    CUFFT_SAFE_CALL(hipfftExecR2C(fPlan, (hipfftReal *)dKernel, (hipfftComplex *)dKernelTransform));

    // Multiply the coefficients together and normalize the result
    ComplexPointwiseMulAndScale<<<32, 256>>>(dSignalTransform, dKernelTransform, n, 1.0f / n);

    // Check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed [ ComplexPointwiseMulAndScale ]");

    // Transform signal back
    CUFFT_SAFE_CALL(hipfftExecC2R(rPlan, (hipfftComplex *)dSignalTransform, (hipfftReal *)dSignal));

    CUDA_SAFE_CALL(hipMemcpy(hSignal, dSignal, memSizeReal,
                              hipMemcpyDeviceToHost));

	
    //Destroy CUFFT context
    CUFFT_SAFE_CALL(hipfftDestroy(fPlan));
	CUFFT_SAFE_CALL(hipfftDestroy(rPlan));

    // cleanup memory
    free(hSignal);
    free(hKernel);
    CUDA_SAFE_CALL(hipFree(dSignal));
    CUDA_SAFE_CALL(hipFree(dKernel));
	CUDA_SAFE_CALL(hipFree(dKernelTransform));
	CUDA_SAFE_CALL(hipFree(dSignalTransform));
}

int main(int argc,char* argv)
{
	CUT_DEVICE_INIT(argc,argv);
	const int n = 256;
	float signal[n]; 	
	float kernel[n];
		
	//printf("the signal: \n");
	
	//initialize signal and filter
	for (unsigned int i = 0; i<n;i++)
	{
		if(i%2==0)
			signal[i] = 1;
		else
			signal[i] = -1;
			
			//printf("%1.1f ", signal[i]);
			
		if(i<(1+n/2))
			kernel[i]=1;
		else
			kernel[i]=0;
			
	}
	
	/*
	printf("\n \n initialize filteredSignal's memory to constant: \n");
	 
	for (unsigned int i = 0; i<n;i++)
		printf("%1.1f ", filteredSignal[i]);
		
	printf("\n \n the kernel: \n");
	
	for (unsigned int i = 0; i<n;i++)
		printf("%1.1f ", kernel[i]);
		
	printf("\n \n the filtered signal: \n");
	*/
	
	//define timing variables
	time_t startTime;
	time_t endTime;
	double runTime;
	double timePerCall;
	int nCalls = 100000;
	
	
	time(&startTime);
	//call the filtering function
	for(unsigned int i = 0; i<nCalls;i++)
	fftFilter( &(signal[0]), &(kernel[0]), n);
	
	time(&endTime);
	printf("start: %ld ", startTime);
	printf("end: %ld ", endTime);
	runTime = difftime(endTime,startTime);
	timePerCall = runTime*(double)1000/(double)nCalls;
	
	
	/*	
	for (unsigned int i = 0; i<n;i++)
		printf("%1.1f ", filteredSignal[i]);
	*/
		
	printf("\n \n total runtime for %i calls was %f seconds", nCalls, runTime);
	printf("\n \n time per call: %f ms" ,timePerCall);
	getchar();
	return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
    Complex c;
    c[0] = a[0] + b[0];
    c[1] = a[1] + b[1];
    return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
    Complex c;
    c[0] = s * a[0];
    c[1] = s * a[1];
    return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c[0] = a[0] * b[0] - a[1] * b[1];
    c[1] = a[0] * b[1] + a[1] * b[0];
    return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex* a, const Complex* b, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);     
} 