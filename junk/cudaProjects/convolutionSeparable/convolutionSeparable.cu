#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  Users and possessors of this source code 
 * are hereby granted a nonexclusive, royalty-free license to use this code 
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as 
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer  software"  and "commercial computer software 
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein. 
 *
 * Any use of this source code in individual and commercial software must 
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/*
 * This sample implements a separable convolution filter 
 * of a 2D signal with a gaussian kernel.
 */



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil.h>



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}


////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
//Global macro, controlling innermost convolution loop unrolling
#define UNROLL_INNER
#include <convolutionSeparable_kernel.cu>



////////////////////////////////////////////////////////////////////////////////
// Data configuration
//////////////////////////////////------//////////////////////////////////////////////
//Image width should be aligned to maximum coalesced read/write size
//for best global memory performance in both row and column filter.
#ifdef __DEVICE_EMULATION__
//Reduce problem size to have reasonable emulation time
const int      DATA_W = iAlignUp(256, 16);
const int      DATA_H = 256;
#else
const int      DATA_W = iAlignUp(8192, 16);
const int      DATA_H = 1024;//CHANGE NUMBER OF ROWS HERE!
#endif
const int   DATA_SIZE = DATA_W * DATA_H * sizeof(float);
const int KERNEL_SIZE = KERNEL_W * sizeof(float);

//////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////// filtering on the gpu wrapped for labview, matlab, etc /////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" __declspec(dllexport) void gpuFilter(float* h_Result,  int dataW, int dataH)
//void gpuFilter(float* h_Result,   float* h_Data,    int dataW, int dataH)
	{
	////////////////////////// kernel init /////////////////////////////////////
	 //declare and allocate for kernel on host
	 float *h_Kernel, kernelSum = 0;
	 h_Kernel    = (float *)malloc(KERNEL_SIZE);
	 
	 //build the kernel
        for(unsigned int i = 0; i < KERNEL_W; i++){
            float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
            h_Kernel[i] = expf(- dist * dist / 2);
            kernelSum += h_Kernel[i];
        }
        for(unsigned int i = 0; i < KERNEL_W; i++)
            h_Kernel[i] /= kernelSum;

	//copy host kernel to device
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE) );
	free(h_Kernel);//free the host kernel memory
	////////////////////////// end kernel init /////////////////////////////////////
	
	////////////////////////////////fitlering procedure/////////////////////////////
	//declare and initialize device variables
	float  *d_Data, *d_Result, *d_Temp;
	
	CUDA_SAFE_CALL( hipMalloc( (void **)&d_Data, DATA_SIZE) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&d_Result, DATA_SIZE) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&d_Temp , DATA_SIZE) );
	
	//transfer the data
	CUDA_SAFE_CALL( hipMemcpy(d_Data, h_Result, DATA_SIZE, hipMemcpyHostToDevice) );
    
	//setup for the call to the device
	dim3 blockGridRows(iDivUp(DATA_W, ROW_TILE_W), DATA_H);
    dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS);
	
	//call the device funtion
	convolutionRowGPU<<<blockGridRows, threadBlockRows>>>( d_Result, d_Data,  dataW, dataH);
	
	//obtain the result
	CUDA_SAFE_CALL( hipMemcpy(h_Result, d_Result, DATA_SIZE, hipMemcpyDeviceToHost) );
	
	//clean up device memory
	
	CUDA_SAFE_CALL( hipFree(d_Data) );
    CUDA_SAFE_CALL( hipFree(d_Result) );
	CUDA_SAFE_CALL( hipFree(d_Temp) );
	//////////////////////////// end filtering procedure ////////////////////////////////
	}
//////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////

int __stdcall DllMain(void)
{
return 0;
}
